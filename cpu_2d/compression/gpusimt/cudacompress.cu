#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

// PFOR and PFOR-DELTA Compression and decompression routines


#include <stdio.h>
#include <iomanip>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/transform.h>
#include <thrust/partition.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/unique.h>
#include <thrust/gather.h>
#include <thrust/sort.h>
#include <thrust/merge.h>
#include <thrust/functional.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <queue>
#include <string>
#include <map>
#include <unordered_map>
#include <set>
#include <vector>
#include <stack>
#include <ctime>
#include <limits>
#include <fstream>
//#include "cm.h"

using namespace std;
using namespace thrust::system::cuda::experimental;

typedef long long int int_type;
typedef unsigned int int32_type;
typedef unsigned short int int16_type;
typedef unsigned char int8_type;
typedef double float_type;

thrust::device_vector<unsigned char> scratch;
size_t int_size;
size_t float_size;
bool delta;
bool phase_copy = 0;
map<string, unsigned int> cpy_bits;
map<string, long long int> cpy_init_val;
map<string, unsigned int> cnt_counts;
string curr_file;
map<string,bool> min_max_eq;

#ifndef gpuErrchk
#define gpuErrchk(ans)
{
    gpuAssert((ans), __FILE__, __LINE__);
}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#endif

template<typename T>
struct type_to_int64
{
    const T *source;
    long long int *dest;
    long long int *ad;
    type_to_int64(const T* _source, long long int *_dest, long long int *_ad):
              source(_source), dest(_dest), ad(_ad) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        dest[i] = (int_type)source[i] + ad[0];
    }
};

template<typename T>
struct int64_to_type
{
    __host__ __device__
    unsigned int operator()(const int_type x)
    {
        return (T)x;
    }
};

template<typename T>
struct to_int64
{
    __host__ __device__
    int_type operator()(const T x)
    {
        return (int_type)x;
    }
};

struct compress_functor_int
{

    const int_type * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;
    compress_functor_int(const int_type * _source, unsigned long long int  * _dest,
                         const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val = source[i] - start_val[0];
        unsigned int shifted = vals[2] - vals[0] - (i%vals[1])*vals[0];
        dest[i] = val << shifted;
    }
};

struct compress_functor_float
{
    const long long int * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;
    compress_functor_float(const long long int * _source, unsigned long long int  * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val;
        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = vals[2];
        val = source[i] - start_val[0];
        unsigned int z = i%fit_count;
        unsigned int shifted = int_sz - bits - z*bits;
        dest[i] = val << shifted;
    }
};

struct decompress_functor_int
{
    const unsigned long long int * source;
    int_type * dest;
    const long long int * start_val;
    const unsigned int * vals;
    decompress_functor_int(const unsigned long long int * _source, int_type * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        unsigned long long int tmp = source[i/vals[1]]  >> (vals[2] - vals[0] - (i%vals[1])*vals[0]);
        // set  the rest of bits to 0
        tmp = tmp << (vals[2] - vals[0]);
        tmp = tmp >> (vals[2] - vals[0]);
        dest[i] = tmp + start_val[0];
    }
};

/**
 * pfor_delta algorithm decompression
 * @param  destination [description]
 * @param  host        [description]
 * @param  d_v         [description]
 * @param  s_v         [description]
 * @param  colname     [description]
 * @return             [description]
 */
size_t pfor_decompress(void* destination, void* host, void* d_v, void* s_v, string colname)
{
    unsigned int bit_count = 64;
    auto cnt = ((unsigned int*)host)[0];
    auto orig_upper_val = ((long long int*)((char*)host +12))[0];
    auto orig_recCount = ((unsigned int*)((char*)host + cnt))[7];
    auto bits = ((unsigned int*)((char*)host + cnt))[8];
    auto orig_lower_val = ((long long int*)((unsigned int*)((char*)host + cnt) + 9))[0];
    auto fit_count = ((unsigned int*)((char*)host + cnt))[11];
    auto start_val = ((long long int*)((unsigned int*)((char*)host + cnt) + 12))[0];
    auto comp_type = ((unsigned int*)host)[5];

    //cout << "Decomp Header " <<  orig_recCount << " " << bits << " " << orig_lower_val << " " << cnt << " " << fit_count << " " << comp_type << endl;
    //cout << colname << " " << orig_lower_val << " " << orig_upper_val << endl;
    if(orig_lower_val == orig_upper_val)
    {
        min_max_eq[colname] = 1;
    } else
    {
        min_max_eq[colname] = 0;
    }
    if(scratch.size() < cnt)
    {
        scratch.resize(cnt);
    }
    hipMemcpy(thrust::raw_pointer_cast(scratch.data()), (void*)((unsigned int*)host + 6), cnt, hipMemcpyHostToDevice);
    thrust::device_ptr<int_type> d_int((int_type*)destination);

    if(comp_type == 1)
    {
        thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
        thrust::device_ptr<long long int> dd_sv((long long int*)s_v);
        dd_sv[0] = orig_lower_val;
        dd_v[0] = bits;
        dd_v[1] = fit_count;
        dd_v[2] = bit_count;
        thrust::counting_iterator<unsigned int> begin(0);
        decompress_functor_int ff1((const unsigned long long int *)thrust::raw_pointer_cast(scratch.data()),(int_type*)destination, (long long int*)s_v, (unsigned int*)d_v);
        thrust::for_each(begin, begin + orig_recCount, ff1);
        d_int[0] = start_val;
        thrust::inclusive_scan(d_int, d_int + orig_recCount, d_int);
    }
    else {
        if(!phase_copy) {
            thrust::device_vector<int_type> ad(1);
            ad[0] = orig_lower_val;
            thrust::counting_iterator<unsigned int> begin(0);
            if(bits == 8)
            {
                type_to_int64<unsigned char> ff1((const unsigned char *)thrust::raw_pointer_cast(scratch.data()),(int_type*)destination, thrust::raw_pointer_cast(ad.data()));
                thrust::for_each(begin, begin + orig_recCount, ff1);
            }
            else if(bits == 16)
            {
                type_to_int64<unsigned short int> ff1((const unsigned short int *)thrust::raw_pointer_cast(scratch.data()),(int_type*)destination, thrust::raw_pointer_cast(ad.data()));
                thrust::for_each(begin, begin + orig_recCount, ff1);
            }
            else if(bits == 32)
            {
                type_to_int64<unsigned int> ff1((const unsigned int *)thrust::raw_pointer_cast(scratch.data()),(int_type*)destination, thrust::raw_pointer_cast(ad.data()));
                thrust::for_each(begin, begin + orig_recCount, ff1);
            }
            else
            {
                type_to_int64<long long int> ff1((const long long int *)thrust::raw_pointer_cast(scratch.data()),(int_type*)destination, thrust::raw_pointer_cast(ad.data()));
                thrust::for_each(begin, begin + orig_recCount, ff1);
            };
        }
        else {
            cpy_bits[colname] = bits;
            cpy_init_val[colname] = orig_lower_val;
            if(bits == 8)
            {
                thrust::device_ptr<unsigned char> dest((unsigned char*)destination);
                thrust::copy(scratch.begin(), scratch.begin()+orig_recCount, dest);
            }
            else if(bits == 16)
            {
                thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(scratch.data()));
                thrust::device_ptr<unsigned short int> dest((unsigned short int*)destination);
                thrust::copy(src, src+orig_recCount, dest);
            }
            else if(bits == 32)
            {
                thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(scratch.data()));
                thrust::device_ptr<unsigned int> dest((unsigned int*)destination);
                thrust::copy(src, src+orig_recCount, dest);
            }
            else
            {
                thrust::device_ptr<int_type> src((int_type*)thrust::raw_pointer_cast(scratch.data()));
                thrust::copy(src, src+orig_recCount, d_int);
            };
        };
    };
    return orig_recCount;
}

/**
 * pfor_delta algorithm compression
 * @param source     [description]
 * @param source_len [description]
 * @param file_name  [description]
 * @param host       [description]
 * @param tp         [description]
 */
template< typename T>
void pfor_delta_compress(void* source, size_t source_len, string file_name, thrust::host_vector<T, pinned_allocator<T> >& host, bool tp)
{
    long long int orig_lower_val, orig_upper_val, start_val, real_lower, real_upper;
    unsigned int  bits, recCount;
    unsigned int bit_count = 8*8;
    unsigned int fit_count;
    unsigned int comp_type = 1; // FOR-DELTA

    if(tp == 0)
    {
        recCount = source_len/int_size;
    } else
    {
        recCount = source_len/float_size;
    }
    void* ss;
    gpuErrchk(hipMalloc((void **) &ss, recCount*float_size));

    if (tp == 0)
    {
        thrust::device_ptr<int_type> s((int_type*)source);
        thrust::device_ptr<int_type> d_ss((int_type*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);

        start_val = d_ss[0];
        if(recCount > 1)
        {
            d_ss[0] = d_ss[1];
        }
        orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
        orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));
        real_lower = s[0];
        real_upper = s[recCount-1];
        //cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
        //cout << "We need for delta " << file_name << " " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " <<  orig_upper_val-orig_lower_val << " values " << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
        if (bits == 0)
        {
            bits = 1;
        }
    }
    else
    {
        thrust::device_ptr<long long int> s((long long int*)source);
        thrust::device_ptr<long long int> d_ss((long long int*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);
        start_val = d_ss[0];
        if(recCount > 1)
        {
            d_ss[0] = d_ss[1];
        }
        orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
        orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));
        real_lower = s[0];
        real_upper = s[recCount-1];

        //cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
        //cout << "We need for delta " << file_name << " " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " << orig_upper_val-orig_lower_val << " values" << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
        if (bits == 0)
        {
            bits = 1;
        }
    };
    thrust::counting_iterator<unsigned int> begin(0);
    fit_count = bit_count/bits;
    void* d_v1;
    gpuErrchk(hipMalloc((void **) &d_v1, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v1);
    void* s_v1;
    gpuErrchk(hipMalloc((void **) &s_v1, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v1);
    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;
    thrust::device_ptr<char> dd((char*)source);
    thrust::fill(dd, dd+source_len,0);
    //cout << "FF " << orig_lower_val << " " << bits << " " << fit_count << " " << bit_count << endl;
    if (tp == 0)
    {
        compress_functor_int ff((int_type*)ss,(unsigned long long int*)source, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    }
    else
    {
        compress_functor_float ff((long long int*)ss,(unsigned long long int*)source, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    };
    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)source);
    // make an addition  sequence
    thrust::device_ptr<unsigned long long int> add_seq((unsigned long long int*)ss);
    thrust::constant_iterator<unsigned long long int> iter(fit_count);
    thrust::sequence(add_seq, add_seq + recCount, 0, 1);
    thrust::transform(add_seq, add_seq + recCount, iter, add_seq, thrust::divides<unsigned long long int>());

    unsigned int cnt = (recCount)/fit_count;
    if (recCount%fit_count > 0)
    {
        ++cnt;
    }
    thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);
    thrust::reduce_by_key(add_seq, add_seq+recCount,s_copy1,thrust::make_discard_iterator(), fin_seq);
    // copy fin_seq to host
    unsigned long long int * raw_src = thrust::raw_pointer_cast(fin_seq);
    //cout << file_name << " CNT  " << cnt << " " << recCount << endl;
    cnt = cnt*8;

    hipMemcpy( host.data(), (void *)raw_src, cnt, hipMemcpyDeviceToHost);
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&real_lower, 8);
    binary_file.write((char *)&real_upper, 8);
    binary_file.write((char *)&comp_type, 4);
    binary_file.write((char *)host.data(),cnt);
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&recCount, 4);
    binary_file.write((char *)&bits, 4);
    binary_file.write((char *)&orig_lower_val, 8);
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&start_val, 8);
    binary_file.close();
    if(cnt_counts[curr_file] < cnt)
    {
            cnt_counts[curr_file] = cnt;
    }

    thrust::device_free(fin_seq);
    hipFree(ss);
    hipFree(d_v1);
    hipFree(s_v1);
}



/**
 * non sorted compressed fields should have 1,2,4 or 8 byte values for direct operations on compressed values
 * @param source     [description]
 * @param source_len [description]
 * @param file_name  [description]
 * @param host       [description]
 * @param tp         [description]
 */
/*
template< typename T>
void pfor_compress(void* source, size_t source_len, string file_name, thrust::host_vector<T, pinned_allocator<T> >& host,  bool tp)
//void pfor_compress(void* source, size_t source_len, string file_name, thrust::host_vector<T>& host,  bool tp)
{
    unsigned int recCount = source_len/int_size;
    long long int orig_lower_val;
    long long int orig_upper_val;
    unsigned int  bits;
    unsigned int fit_count = 0;
    unsigned int comp_type = 0; // FOR
    long long int start_val = 0;
    bool sorted = 0;

    // check if sorted

    if(delta) {
        if (tp == 0) {
            thrust::device_ptr<int_type> s((int_type*)source);
            sorted = thrust::is_sorted(s, s+recCount);
        }
        else {
            recCount = source_len/float_size;
            thrust::device_ptr<long long int> s((long long int*)source);
            sorted = thrust::is_sorted(s, s+recCount);
        };
        //cout << "file " << file_name << " is sorted " << sorted << endl;

        if(sorted) {
            pfor_delta_compress(source, source_len, file_name, host, tp);
            return;
        };
    };


    //cout << "Recs " << recCount << endl;
    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);
        orig_lower_val = *(thrust::min_element(s, s + recCount));
        orig_upper_val = *(thrust::max_element(s, s + recCount));
        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    }
    else {

        thrust::device_ptr<long long int> s((long long int*)source);
        orig_lower_val = *(thrust::min_element(s, s + recCount));
        orig_upper_val = *(thrust::max_element(s, s + recCount));
        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    };

    if (bits != 8 && bits != 16 && bits != 32 && bits != 64) {
        if(bits < 8)
            bits = 8;
        else if(bits < 16)
            bits = 16;
        else if(bits < 32)
            bits = 32;
        else if(bits < 64)
            bits = 64;
    };
    //cout << "We will really need " << bits << " for " << file_name << endl;

    unsigned int cnt;
    thrust::device_ptr<int_type> s((int_type*)source);
    thrust::constant_iterator<int_type> iter(orig_lower_val);
    thrust::transform(s, s+recCount, iter, s, thrust::minus<int_type>());

    thrust::device_vector<int8_type> d_columns_int8;
    thrust::device_vector<int16_type> d_columns_int16;
    thrust::device_vector<int32_type> d_columns_int32;
    if(bits == 8) {
        d_columns_int8.resize(recCount);
        thrust::transform(s, s+recCount, d_columns_int8.begin(), int64_to_type<int8_type>());
        hipMemcpy( host.data(), thrust::raw_pointer_cast(d_columns_int8.data()), recCount, hipMemcpyDeviceToHost);
        cnt = recCount;
    }
    else if(bits == 16) {
        d_columns_int16.resize(recCount);
        thrust::transform(s, s+recCount, d_columns_int16.begin(), int64_to_type<int16_type>());
        hipMemcpy( host.data(), thrust::raw_pointer_cast(d_columns_int16.data()), recCount*2, hipMemcpyDeviceToHost);
        cnt = recCount*2;
    }
    else if(bits == 32) {
        d_columns_int32.resize(recCount);
        thrust::transform(s, s+recCount, d_columns_int32.begin(), int64_to_type<int32_type>());
        hipMemcpy( host.data(), thrust::raw_pointer_cast(d_columns_int32.data()), recCount*4, hipMemcpyDeviceToHost);
        cnt = recCount*4;
    }
    else {
        hipMemcpy( host.data(), (void*)source, recCount*8, hipMemcpyDeviceToHost);
        cnt = recCount*8;
    };

    fit_count = 64/bits;


 //cout << "comp Header " <<  file_name << " " << recCount << " " << bits << " " << orig_lower_val << " " << cnt << " " << fit_count << " " << comp_type << " " << orig_upper_val << " " << start_val <<  endl;
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&orig_lower_val, 8);
    binary_file.write((char *)&orig_upper_val, 8);
    binary_file.write((char *)&comp_type, 4);
    binary_file.write((char *)host.data(),cnt);
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&recCount, 4);
    binary_file.write((char *)&bits, 4);
    binary_file.write((char *)&orig_lower_val, 8);
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&start_val, 8);
    binary_file.close();
    if(cnt_counts[curr_file] < cnt)
        cnt_counts[curr_file] = cnt;

}
*/


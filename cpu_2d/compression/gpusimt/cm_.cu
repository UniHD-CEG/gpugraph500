#include "hip/hip_runtime.h"
/*
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <cctype>
#include <algorithm>
#include <functional>
#include <numeric>
#include "cm.h"
#include "compress.cu"

/*
#include "atof.h"
#include "compress.cu"
#include "sorts.cu"
#include "filter.h"
#include "callbacks.h"
#include "zone_map.h"
*/
#include <unistd.h>

using namespace std;
using namespace thrust::placeholders;

size_t total_count = 0, total_max;
clock_t tot;
unsigned int total_segments = 0;
size_t process_count;
size_t alloced_sz = 0;
bool fact_file_loaded = 1;
bool verbose;
bool interactive, ssd, delta, star;
void* d_v = nullptr;
void* s_v = nullptr;
queue<string> op_sort;
queue<string> op_presort;
queue<string> op_type;
bool op_case = 0;
queue<string> op_value;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<unsigned int> op_nums_precision;
queue<string> col_aliases;
//map<string, map<string, col_data> > data_dict;
map<unsigned int, map<unsigned long long int, size_t> > char_hash;

map<string, char*> index_buffers;
map<string, char*> buffers;
map<string, size_t> buffer_sizes;
size_t total_buffer_size;
queue<string> buffer_names;

void* alloced_tmp;
bool alloced_switch = 0;
//map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
map<string, unsigned int> cpy_bits;
map<string, long long int> cpy_init_val;
char* readbuff = nullptr;


struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON) || ((x-y) < -EPSILON);
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};

template <typename T>
struct power_functor : public thrust::unary_function<T,T>
{
    unsigned int a;

    __host__ __device__
    power_functor(unsigned int a_) { a = a_; }

    __host__ __device__
    T operator()(T x)
    {
		return x*(unsigned int)pow((double)10,(double)a);
	}
};

/*class power_functor {

    unsigned int a;

    public:

        power_functor(unsigned int a_) { a = a_; }

        __host__ __device__ int_type operator()(int_type x) const
        {
            return x*(unsigned int)pow((double)10,(double)a);
        }
};
*/


/*
void allocColumns(CudaSet* a, queue<string> fields);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt);
void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void write_compressed_char(string file_name, unsigned int index, size_t mCount);
size_t getFreeMem();
size_t getTotalSystemMemory();
void process_error(int severity, string err);
*/





































/*
int_type reverse_op(int_type op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
    size_t available, total;
    hipMemGetInfo(&available, &total);
    return available;
} ;



void allocColumns(CudaSet* a, queue<string> fields)
{
    if(a->filtered) {
        CudaSet* t;
        if(a->filtered)
            t = varNames[a->source_name];
        else
            t = a;

        if(int_size*t->maxRecs > alloced_sz) {
            if(alloced_sz) {
                hipFree(alloced_tmp);
            };
            hipMalloc((void **) &alloced_tmp, int_size*t->maxRecs);
            alloced_sz = int_size*t->maxRecs;
        }
    }
    else {
        while(!fields.empty()) {
            if(var_exists(a, fields.front()) && !a->onDevice(fields.front())) {
                a->allocColumnOnDevice(fields.front(), a->maxRecs);
            }
            fields.pop();
        };
    };
}



void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment, size_t& count)
{
    if(!a->onDevice(field)) {
        a->allocColumnOnDevice(field, a->maxRecs);
    };
    if(a->prm_index == 'R') {
        mygather(field, a, t, count, a->mRecCount);
    }
    else {
        mycopy(field, a, t, count, t->mRecCount);
        a->mRecCount = t->mRecCount;
    };
}


void copyFinalize(CudaSet* a, queue<string> fields)
{
	set<string> uniques;
	if(scratch.size() < a->mRecCount*8)
		scratch.resize(a->mRecCount*8);
	thrust::device_ptr<int_type> tmp((int_type*)thrust::raw_pointer_cast(scratch.data()));

	while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()) && cpy_bits.find(fields.front()) != cpy_bits.end())	{

			if(cpy_bits[fields.front()] == 8) {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<char> src((char*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<char>());
				}
				else {
					thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned char>());
				};
			}
			else if(cpy_bits[fields.front()] == 16) {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned short>());
				}
				else {
					thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned short>());
				};
			}
			else if(cpy_bits[fields.front()] == 32) {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned int>());
				}
				else {
					thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned int>());
				};
			}
			else {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<int_type> src((int_type*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::copy(src, src+a->mRecCount, tmp);
				}
				else {
					thrust::device_ptr<int_type> src((int_type*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::copy(src, src+a->mRecCount, tmp);
				};
			};
			thrust::constant_iterator<int_type> iter(cpy_init_val[fields.front()]);
			if(a->type[fields.front()] != 1) {
				thrust::transform(tmp, tmp + a->mRecCount, iter, a->d_columns_int[fields.front()].begin(), thrust::plus<int_type>());
			}
			else {
				thrust::device_ptr<int_type> dest((int_type*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
				thrust::transform(tmp, tmp + a->mRecCount, iter, dest, thrust::plus<int_type>());
                thrust::transform(dest, dest+a->mRecCount, a->d_columns_float[fields.front()].begin(), long_to_float());
			};
		};
		uniques.insert(fields.front());
        fields.pop();
    };
}


void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt)
{
    set<string> uniques;
    if(a->filtered) { //filter the segment
        if(flt) {
            filter_op(a->fil_s, a->fil_f, segment);
        };
        if(rsz && a->mRecCount) {
            queue<string> fields1(fields);
            while(!fields1.empty()) {
                a->resizeDeviceColumn(a->devRecCount + a->mRecCount, fields1.front());
                fields1.pop();
            };
            a->devRecCount = a->devRecCount + a->mRecCount;
        };
    };
	cpy_bits.clear();
	cpy_init_val.clear();
	auto f(fields);

    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()))	{
            if(a->filtered) {
                if(a->mRecCount) {
                    CudaSet *t = varNames[a->source_name];
                    alloced_switch = 1;
                    t->CopyColumnToGpu(fields.front(), segment);
                    gatherColumns(a, t, fields.front(), segment, count);
                    alloced_switch = 0;
                };
            }
            else {
                if(a->mRecCount) {
                    a->CopyColumnToGpu(fields.front(), segment, count);
                };
            };
            uniques.insert(fields.front());
        };
        fields.pop();
    };
}


void mygather(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[colname] != 1 ) {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<unsigned short int> d_col_source((unsigned short int*)alloced_tmp);
					thrust::device_ptr<unsigned short int> d_col_dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_int[colname].begin() + offset);
			};
		}
		else {
			thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
			thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_int[colname].begin() + offset);
		};

    }
    else  {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<unsigned short int> d_col_source((unsigned short int*)alloced_tmp);
					thrust::device_ptr<unsigned short int> d_col_dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_float[colname].begin() + offset);
			};
		}
		else {
			thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
			thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_float[colname].begin() + offset);
		};
    }
};


void mycopy(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[colname] != 1) {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<short int> d_col_source((short int*)alloced_tmp);
					thrust::device_ptr<short int> d_col_dest((short int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()+offset));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col_source((int_type*)alloced_tmp);
					thrust::copy(d_col_source, d_col_source + g_size, a->d_columns_int[colname].begin() + offset);
			};
		}
		else {
			thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
			thrust::copy(d_col, d_col + g_size, a->d_columns_int[colname].begin() + offset);
		};
    }
    else {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<short int> d_col_source((short int*)alloced_tmp);
					thrust::device_ptr<short int> d_col_dest((short int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()+offset));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col_source((int_type*)alloced_tmp);
					thrust::copy(d_col_source, d_col_source + g_size, a->d_columns_float[colname].begin() + offset);
			};
		}
		else {
			thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
			thrust::copy(d_col, d_col + g_size,	a->d_columns_float[colname].begin() + offset);
		};
	};
};



size_t load_queue(queue<string> c1, CudaSet* right, string f2, size_t &rcount,
                  unsigned int start_segment, unsigned int end_segment, bool rsz, bool flt)
{
    queue<string> cc;
    while(!c1.empty()) {
        if(std::find(right->columnNames.begin(), right->columnNames.end(), c1.front()) !=  right->columnNames.end()) {
            if(f2 != c1.front() ) {
                cc.push(c1.front());
            };
        };
        c1.pop();
    };
    if(std::find(right->columnNames.begin(), right->columnNames.end(), f2) !=  right->columnNames.end()) {
        cc.push(f2);
    };

    if(right->filtered) {
        allocColumns(right, cc);
    };

    rcount = right->maxRecs;
    queue<string> ct(cc);

    while(!ct.empty()) {
        if(right->filtered && rsz) {
            right->mRecCount = 0;
        }
        else {
            right->allocColumnOnDevice(ct.front(), rcount*right->segCount);
        };
        ct.pop();
    };

    size_t cnt_r = 0;
    right->devRecCount = 0;
    for(unsigned int i = start_segment; i < end_segment; i++) {
        if(!right->filtered)
            copyColumns(right, cc, i, cnt_r, rsz, 0);
        else
            copyColumns(right, cc, i, cnt_r, rsz, flt);
        cnt_r = cnt_r + right->mRecCount;
    };

    right->mRecCount = cnt_r;
    return cnt_r;

}

size_t max_char(CudaSet* a)
{
    size_t max_char1 = 8;
    for(unsigned int i = 0; i < a->columnNames.size(); i++) {
        if(a->type[a->columnNames[i]] == 2) {
            if (a->char_size[a->columnNames[i]] > max_char1)
                max_char1 = a->char_size[a->columnNames[i]];
        }
        else if(a->type[a->columnNames[i]] == 0 && a->string_map.find(a->columnNames[i]) != a->string_map.end()) {
            auto s = a->string_map[a->columnNames[i]];
            auto pos = s.find_first_of(".");
            auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
            if (len > max_char1)
                max_char1 = len;
        };
    };
    return max_char1;
};


size_t max_char(CudaSet* a, queue<string> field_names)
{
    size_t max_char = 8;
    while (!field_names.empty()) {
        if (a->type[field_names.front()] == 2) {
            if (a->char_size[field_names.front()] > max_char)
                max_char = a->char_size[field_names.front()];
        };
        field_names.pop();
    };
    return max_char;
};


void setSegments(CudaSet* a, queue<string> cols)
{
    size_t mem_available = getFreeMem();
    size_t tot_sz = 0;
    while(!cols.empty()) {
        if(a->type[cols.front()] != 2)
            tot_sz = tot_sz + int_size;
        else
            tot_sz = tot_sz + a->char_size[cols.front()];
        cols.pop();
    };
    if(a->mRecCount*tot_sz > mem_available/3) { //default is 3
        a->segCount = (a->mRecCount*tot_sz)/(mem_available/5) + 1;
        a->maxRecs = (a->mRecCount/a->segCount)+1;
    };
};


void update_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);

    if (SortType.compare("DESC") == 0 )
        str_sort_host(tmp, RecCount, permutation, 1, len);
    else
        str_sort_host(tmp, RecCount, permutation, 0, len);
}


void apply_permutation_char(char* key, unsigned int* permutation, size_t RecCount, char* tmp, unsigned int len)
{
    // copy keys to temporary vector
    hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);
    // permute the keys
    str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, char* res, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}


void filter_op(const char *s, const char *f, unsigned int segment)
{
    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
    //std::clock_t start1 = std::clock();

    if(a->mRecCount == 0 && !a->filtered) {
        b = new CudaSet(0,1);
    }
    else {
        if(verbose)
            cout << "FILTER " << s << " " << f << " " << getFreeMem() << '\xd';

        b = varNames[s];
        b->name = s;
        b->string_map = a->string_map;
        size_t cnt = 0;
        allocColumns(a, b->fil_value);

        if (b->prm_d.size() == 0) {
            b->prm_d.resize(a->maxRecs);
		};

        //cout << endl << "MAP CHECK start " << segment <<  endl;
        char map_check = zone_map_check(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, b->fil_nums_precision, a, segment);
        //cout << endl << "MAP CHECK segment " << segment << " " << map_check <<  endl;

        if(map_check == 'R') {
			auto old_ph = phase_copy;
			phase_copy = 0;
            copyColumns(a, b->fil_value, segment, cnt);
			phase_copy = old_ph;
            bool* res = filter(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, b->fil_nums_precision, a, segment);
            thrust::device_ptr<bool> bp((bool*)res);
            b->prm_index = 'R';
            b->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 1);
            thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
                            bp, b->prm_d.begin(), thrust::identity<bool>());
            hipFree(res);
        }
        else  {
			b->prm_index = map_check;
			if(map_check == 'A')
				b->mRecCount = a->mRecCount;
			else
				b->mRecCount = 0;
        };
        if(segment == a->segCount-1)
            a->deAllocOnDevice();
    }
    if(verbose)
        cout << endl << "filter res " << b->mRecCount << " " << phase_copy << endl;
}




size_t load_right(CudaSet* right, string f2, queue<string> op_g, queue<string> op_alt, size_t& rcount, unsigned int start_seg, unsigned int end_seg) {

    size_t cnt_r = 0;
    //if join is on strings then add integer columns to left and right tables and modify colInd1 and colInd2

    // need to allocate all right columns
    if(right->not_compressed) {
        queue<string> op_alt1;
        op_alt1.push(f2);
        cnt_r = load_queue(op_alt1, right, "", rcount, start_seg, end_seg, 1, 1);

        queue<string> op_alt2;
        while(!op_alt.empty()) {
            if(f2.compare(op_alt.front())) {
                if (std::find(right->columnNames.begin(), right->columnNames.end(), op_alt.front()) != right->columnNames.end()) {
                    op_alt2.push(op_alt.front());
                };
            };
            op_alt.pop();
        };
        if(!op_alt2.empty())
            cnt_r = load_queue(op_alt2, right, "", rcount, start_seg, end_seg, 0, 0);
    }
    else {
        cnt_r = load_queue(op_alt, right, f2, rcount, start_seg, end_seg, 1, 1);
    };

    return cnt_r;
};



void insert_records(const char* f, const char* s) {
    char buf[4096];
    size_t size, maxRecs, cnt = 0;
    string str_s, str_d;

    if(varNames.find(s) == varNames.end()) {
        process_error(3, "couldn't find " + string(s) );
    };
    CudaSet *a;
    a = varNames.find(s)->second;
    a->name = s;

    if(varNames.find(f) == varNames.end()) {
        process_error(3, "couldn't find " + string(f) );
    };

    CudaSet *b;
    b = varNames.find(f)->second;
    b->name = f;

    // if both source and destination are on disk
    cout << "SOURCES " << a->source << ":" << b->source << endl;
    if(a->source && b->source) {
        for(unsigned int i = 0; i < a->segCount; i++) {
            for(unsigned int z = 0; z < a->columnNames.size(); z++) {

				if(a->type[a->columnNames[z]] != 2) {
					str_s = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i);
					str_d = b->load_file_name + "." + a->columnNames[z] + "." + to_string(b->segCount + i);
					cout << str_s << " " << str_d << endl;
					FILE* source = fopen(str_s.c_str(), "rb");
					FILE* dest = fopen(str_d.c_str(), "wb");
					while (size = fread(buf, 1, BUFSIZ, source)) {
						fwrite(buf, 1, size, dest);
					}
					fclose(source);
					fclose(dest);
				}
				else { //merge strings
					//read b's strings
					str_s = b->load_file_name + "." + b->columnNames[z];
					FILE* dest = fopen(str_s.c_str(), "rb");
					auto len = b->char_size[b->columnNames[z]];
					map<string, unsigned long long int> map_d;
					buf[len] = 0;
					unsigned long long cnt = 0;
					while (fread(buf, len, 1, dest)) {
						map_d[buf] = cnt;
						cnt++;
					};
					fclose(dest);
					unsigned long long int cct = cnt;

					str_s = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i) + ".hash";
					str_d = b->load_file_name + "." + b->columnNames[z] + "." + to_string(b->segCount + i) + ".hash";
					FILE* source = fopen(str_s.c_str(), "rb");
					dest = fopen(str_d.c_str(), "wb");
					while (size = fread(buf, 1, BUFSIZ, source)) {
						fwrite(buf, 1, size, dest);
					}
					fclose(source);
					fclose(dest);

					str_s = a->load_file_name + "." + a->columnNames[z];
					source = fopen(str_s.c_str(), "rb");
					map<unsigned long long int, string> map_s;
					buf[len] = 0;
					cnt = 0;
					while (fread(buf, len, 1, source)) {
						map_s[cnt] = buf;
						cnt++;
					};
					fclose(source);

					queue<string> op_vx;
					op_vx.push(a->columnNames[z]);
					allocColumns(a, op_vx);
					a->resize(a->maxRecs);
					a->CopyColumnToGpu(a->columnNames[z], z, 0);
					a->CopyColumnToHost(a->columnNames[z]);

					str_d = b->load_file_name + "." + b->columnNames[z];
                    fstream f_file;
                    f_file.open(str_d.c_str(), ios::out|ios::app|ios::binary);

					for(auto j = 0; j < a->mRecCount; j++) {
						auto ss = map_s[a->h_columns_int[a->columnNames[z]][j]];
						if(map_d.find(ss) == map_d.end()) { //add
							f_file.write((char *)ss.c_str(), len);
							a->h_columns_int[a->columnNames[z]][j] = cct;
							cct++;
						}
						else {
							a->h_columns_int[a->columnNames[z]][j] = map_d[ss];
						};
					};
					f_file.close();

					thrust::device_vector<int_type> d_col(a->mRecCount);
					thrust::copy(a->h_columns_int[a->columnNames[z]].begin(), a->h_columns_int[a->columnNames[z]].begin() + a->mRecCount, d_col.begin());
					auto i_name = b->load_file_name + "." + b->columnNames[z] + "." + to_string(b->segCount + i) + ".idx";
					pfor_compress(thrust::raw_pointer_cast(d_col.data()), a->mRecCount*int_size, i_name, a->h_columns_int[a->columnNames[z]], 0);
				};
            };
        };

        if(a->maxRecs > b->maxRecs)
            maxRecs = a->maxRecs;
        else
            maxRecs = b->maxRecs;

        for(unsigned int i = 0; i < b->columnNames.size(); i++) {
            b->reWriteHeader(b->load_file_name, b->columnNames[i], a->segCount + b->segCount, a->totalRecs + b->totalRecs, maxRecs);
        };
    }
    else if(!a->source && !b->source) { //if both source and destination are in memory
        size_t oldCount = b->mRecCount;
        b->resize(a->mRecCount);
        for(unsigned int z = 0; z< b->mColumnCount; z++) {
            if(b->type[a->columnNames[z]] == 0) {
                thrust::copy(a->h_columns_int[a->columnNames[z]].begin(), a->h_columns_int[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_int[b->columnNames[z]].begin() + oldCount);
            }
            else if(b->type[a->columnNames[z]] == 1) {
                thrust::copy(a->h_columns_float[a->columnNames[z]].begin(), a->h_columns_float[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_float[b->columnNames[z]].begin() + oldCount);
            }
            else {
                hipMemcpy(b->h_columns_char[b->columnNames[z]] + b->char_size[b->columnNames[z]]*oldCount, a->h_columns_char[a->columnNames[z]], a->char_size[a->columnNames[z]]*a->mRecCount, hipMemcpyHostToHost);
            };
        };
    }
    else if(!a->source && b->source) {

        total_segments = b->segCount;
        total_count = b->mRecCount;
        total_max = b->maxRecs;;

        queue<string> op_vx;
        for(unsigned int i=0; i < a->columnNames.size(); i++)
            op_vx.push(a->columnNames[i]);

        allocColumns(a, op_vx);
        a->resize(a->maxRecs);
        for(unsigned int i = 0; i < a->segCount; i++) {
            if (a->filtered) {
                copyColumns(a, op_vx, i, cnt);
                a->CopyToHost(0, a->mRecCount);
            };
            a->compress(b->load_file_name, 0, 1, i - (a->segCount-1), a->mRecCount);
        };
        for(unsigned int i = 0; i < b->columnNames.size(); i++) {
            b->writeHeader(b->load_file_name, b->columnNames[i], total_segments);
        };
    };
};



void delete_records(const char* f) {

    CudaSet *a;
    a = varNames.find(f)->second;
    a->name = f;
    size_t totalRemoved = 0;
    size_t maxRecs = 0;

    if(!a->keep) { // temporary variable
        process_error(2, "Delete operator is only applicable to disk based sets\nfor deleting records from derived sets please use filter operator ");
    }
    else {  // read matching segments, delete, compress and write on a disk replacing the original segments

        string str, str_old;
        queue<string> op_vx;
        size_t cnt;
        for ( auto it=data_dict[a->load_file_name].begin() ; it != data_dict[a->load_file_name].end(); ++it ) {
            op_vx.push((*it).first);
            if (std::find(a->columnNames.begin(), a->columnNames.end(), (*it).first) == a->columnNames.end()) {

                if ((*it).second.col_type == 0) {
                    a->type[(*it).first] = 0;
                    a->decimal[(*it).first] = 0;
                    //a->h_columns_int[(*it).first] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    a->h_columns_int[(*it).first] = thrust::host_vector<int_type>();
                    a->d_columns_int[(*it).first] = thrust::device_vector<int_type>();
                }
                else if((*it).second.col_type == 1) {
                    a->type[(*it).first] = 1;
                    a->decimal[(*it).first] = 0;
                    //a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    a->h_columns_float[(*it).first] = thrust::host_vector<float_type>();
                    a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
                }
                else if ((*it).second.col_type == 3) {
                    a->type[(*it).first] = 1;
                    a->decimal[(*it).first] = 1;
                    //a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    a->h_columns_float[(*it).first] = thrust::host_vector<float_type>();
                    a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
                }
                else {
                    a->type[(*it).first] = 2;
                    a->decimal[(*it).first] = 0;
                    a->h_columns_char[(*it).first] = nullptr;
                    a->d_columns_char[(*it).first] = nullptr;
                    a->char_size[(*it).first] = (*it).second.col_length;
                };
                a->columnNames.push_back((*it).first);
            }
        };

        allocColumns(a, op_vx);
        a->resize(a->maxRecs);
        a->prm_d.resize(a->maxRecs);
        size_t cc = a->mRecCount;
        size_t tmp;

        void* d;
        CUDA_SAFE_CALL(hipMalloc((void **) &d, a->maxRecs*float_size));
        unsigned int new_seg_count = 0;
        char map_check;

        for(unsigned int i = 0; i < a->segCount; i++) {

            map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, op_nums_precision, a, i);
            if(verbose)
                cout << "MAP CHECK segment " << i << " " << map_check <<  endl;
            if(map_check != 'N') {

                cnt = 0;
                copyColumns(a, op_vx, i, cnt);
                tmp = a->mRecCount;

                if(a->mRecCount) {
                    bool* res = filter(op_type,op_value,op_nums, op_nums_f, op_nums_precision, a, i);
                    thrust::device_ptr<bool> bp((bool*)res);
                    thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
                                    bp, a->prm_d.begin(), thrust::logical_not<bool>());

                    a->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 0);
                    hipFree(res);

//					cout << "Remained recs count " << a->mRecCount << endl;
                    if(a->mRecCount > maxRecs)
                        maxRecs = a->mRecCount;

                    if (a->mRecCount) {

                        totalRemoved = totalRemoved + (tmp - a->mRecCount);
                        if (a->mRecCount == tmp) { //none deleted
                            if(new_seg_count != i) {
                                for (auto it=data_dict[a->load_file_name].begin() ; it != data_dict[a->load_file_name].end(); ++it ) {
                                    auto colname = (*it).first;
                                    str_old = a->load_file_name + "." + colname + "." + to_string(i);
                                    str = a->load_file_name + "." + colname + "." + to_string(new_seg_count);
                                    remove(str.c_str());
                                    rename(str_old.c_str(), str.c_str());
                                };
                            };
                            new_seg_count++;

                        }
                        else { //some deleted
                            //cout << "writing segment " << new_seg_count << endl;

                            map<string, col_data> s = data_dict[a->load_file_name];
                            for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
                                string colname = (*it).first;
                                str = a->load_file_name + "." + colname + "." + to_string(new_seg_count);

                                if(a->type[colname] == 0) {
                                    thrust::device_ptr<int_type> d_col((int_type*)d);
                                    thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[colname].begin(), d_col);
                                    pfor_compress( d, a->mRecCount*int_size, str, a->h_columns_int[colname], 0);
                                }
                                else if(a->type[colname] == 1) {
                                    thrust::device_ptr<float_type> d_col((float_type*)d);
                                    if(a->decimal[colname]) {
                                        thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
                                        thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                                        thrust::transform(d_col,d_col+a->mRecCount, d_col_dec, float_to_long());
                                        pfor_compress( d, a->mRecCount*float_size, str, a->h_columns_float[colname], 1);
                                    }
                                    else {
                                        thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
                                        thrust::copy(d_col, d_col + a->mRecCount, a->h_columns_float[colname].begin());
                                        fstream binary_file(str.c_str(),ios::out|ios::binary);
                                        binary_file.write((char *)&a->mRecCount, 4);
                                        binary_file.write((char *)(a->h_columns_float[colname].data()),a->mRecCount*float_size);
                                        unsigned int comp_type = 3;
                                        binary_file.write((char *)&comp_type, 4);
                                        binary_file.close();

                                    };
                                }
                                else {
                                    thrust::device_ptr<int_type> d_col((int_type*)d);
                                    thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[colname].begin(), d_col);
                                    pfor_compress( d, a->mRecCount*int_size, str + ".hash", a->h_columns_int[colname], 0);
                                };
                            };
                            new_seg_count++;
                        };
                    }
                    else {
                        totalRemoved = totalRemoved + tmp;
                    };
                }
            }
            else {
                if(new_seg_count != i) {
                    for(unsigned int z = 0; z < a->columnNames.size(); z++) {
                        str_old = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i);
                        str = a->load_file_name + "." + a->columnNames[z] + "." + to_string(new_seg_count);
                        remove(str.c_str());
                        rename(str_old.c_str(), str.c_str());
                    };
                };
                new_seg_count++;
                maxRecs	= a->maxRecs;
            };
        };

        if (new_seg_count < a->segCount) {
            for(unsigned int i = new_seg_count; i < a->segCount; i++) {
                //cout << "delete segment " << i << endl;
                for(unsigned int z = 0; z < a->columnNames.size(); z++) {
                    str = a->load_file_name + "." + a->columnNames[z];
                    str += "." + to_string(i);
                    remove(str.c_str());
                };
            };
        };

        for(unsigned int i = new_seg_count; i < a->segCount; i++) {
            a->reWriteHeader(a->load_file_name, a->columnNames[i], new_seg_count, a->totalRecs-totalRemoved, maxRecs);
        };


        a->mRecCount = cc;
        a->prm_d.resize(0);
        a->segCount = new_seg_count;
        a->deAllocOnDevice();
        hipFree(d);
    };


};


void save_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
    size_t str_len;
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    size_t len = data_dict.size();
    binary_file.write((char *)&len, 8);
    for (auto it=data_dict.begin() ; it != data_dict.end(); ++it ) {
        str_len = (*it).first.size();
        binary_file.write((char *)&str_len, 8);
        binary_file.write((char *)(*it).first.data(), str_len);
        map<string, col_data> s = (*it).second;
        size_t len1 = s.size();
        binary_file.write((char *)&len1, 8);

        for (auto sit=s.begin() ; sit != s.end(); ++sit ) {
            str_len = (*sit).first.size();
            binary_file.write((char *)&str_len, 8);
            binary_file.write((char *)(*sit).first.data(), str_len);
            binary_file.write((char *)&(*sit).second.col_type, 4);
            binary_file.write((char *)&(*sit).second.col_length, 4);
        };
    };
    binary_file.close();
}

void load_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
    size_t str_len, recs, len1;
    string str1, str2;
    char buffer[4000];
    unsigned int col_type, col_length;
    fstream binary_file;
    binary_file.open(file_name.c_str(),ios::in|ios::binary);
    if(binary_file.is_open()) {
        binary_file.read((char*)&recs, 8);
        for(unsigned int i = 0; i < recs; i++) {
            binary_file.read((char*)&str_len, 8);
            binary_file.read(buffer, str_len);
            str1.assign(buffer, str_len);
            binary_file.read((char*)&len1, 8);

            for(unsigned int j = 0; j < len1; j++) {
                binary_file.read((char*)&str_len, 8);
                binary_file.read(buffer, str_len);
                str2.assign(buffer, str_len);
                binary_file.read((char*)&col_type, 4);
                binary_file.read((char*)&col_length, 4);
                data_dict[str1][str2].col_type = col_type;
                data_dict[str1][str2].col_length = col_length;
                //cout << "data DICT " << str1 << " " << str2 << " " << col_type << " " << col_length << endl;
            };
        };
        binary_file.close();
    }
    else {
        cout << "Couldn't open data dictionary" << endl;
    };
}

bool var_exists(CudaSet* a, string name) {

    if(std::find(a->columnNames.begin(), a->columnNames.end(), name) !=  a->columnNames.end())
        return 1;
    else

        return 0;
}

int file_exist (const char *filename)
{
    std::ifstream infile(filename);
    return infile.good();
}

bool check_bitmap_file_exist(CudaSet* left, CudaSet* right)
{
    queue<string> cols(right->fil_value);
    bool bitmaps_exist = 1;

    if(cols.size() == 0) {
        bitmaps_exist = 0;
    };
    while(cols.size() ) {
        if (std::find(right->columnNames.begin(), right->columnNames.end(), cols.front()) != right->columnNames.end()) {
            string fname = left->load_file_name + "."  + right->load_file_name + "." + cols.front() + ".0";
            if( !file_exist(fname.c_str())) {
                bitmaps_exist = 0;
            };
        };
        cols.pop();
    };
    return bitmaps_exist;
}

bool check_bitmaps_exist(CudaSet* left, CudaSet* right)
{
    //check if there are join bitmap indexes
    queue<string> cols(right->fil_value);
    bool bitmaps_exist = 1;

    if(cols.size() == 0) {
        bitmaps_exist = 1;
        return 1;
    };
    while(cols.size() ) {
        if (std::find(right->columnNames.begin(), right->columnNames.end(), cols.front()) != right->columnNames.end()) {
            string fname = left->load_file_name + "."  + right->load_file_name + "." + cols.front() + ".0";
            if( !file_exist(fname.c_str())) {
                bitmaps_exist = 0;
            };
        };
        cols.pop();
    };
    if(bitmaps_exist) {
        while(!right->fil_nums.empty() ) {
            left->fil_nums.push(right->fil_nums.front());
            right->fil_nums.pop();
        };
        while(!right->fil_nums_f.empty() ) {
            left->fil_nums_f.push(right->fil_nums_f.front());
            right->fil_nums_f.pop();
        };
        while(!right->fil_value.empty() ) {
            if (std::find(right->columnNames.begin(), right->columnNames.end(), right->fil_value.front()) != right->columnNames.end()) {
                string fname = left->load_file_name + "."  + right->load_file_name + "." + right->fil_value.front();
                left->fil_value.push(fname);
            }
            else
                left->fil_value.push(right->fil_value.front());
            right->fil_value.pop();
        };
        bool add_and = 1;
        if(left->fil_type.empty())
            add_and = 0;
        while(!right->fil_type.empty() ) {
            left->fil_type.push(right->fil_type.front());
            right->fil_type.pop();
        };
        if(add_and) {
            left->fil_type.push("AND");
        };
        return 1;
    }
    else {
        return 0;
    };
}


void check_sort(const string str, const char* rtable, const char* rid)
{
    CudaSet* right = varNames.find(rtable)->second;
    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::app);
    binary_file.write((char *)&right->sort_check, 1);
    binary_file.close();
}

void update_char_permutation(CudaSet* a, string colname, unsigned int* raw_ptr, string ord, void* temp, bool host)
{
    auto s = a->string_map[colname];
    auto pos = s.find_first_of(".");
    auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;

    a->h_columns_char[colname] = new char[a->mRecCount*len];
    memset(a->h_columns_char[colname], 0, a->mRecCount*len);

	thrust::device_ptr<unsigned int> perm(raw_ptr);
	thrust::device_ptr<int_type> temp_int((int_type*)temp);
	thrust::gather(perm, perm+a->mRecCount, a->d_columns_int[colname].begin(), temp_int);

	//for(int z = 0 ; z < a->mRecCount; z++) {
	//cout << "Init vals " << a->d_columns_int[colname][z] << " " << perm[z] << " " << temp_int[z] << endl;
	//};

	//cout << "sz " << a->h_columns_int[colname].size() << " " << a->d_columns_int[colname].size() <<  " " << len << endl;
	hipMemcpy(thrust::raw_pointer_cast(a->h_columns_int[colname].data()), temp, 8*a->mRecCount, hipMemcpyDeviceToHost);

    FILE *f;
    f = fopen(a->string_map[colname].c_str(), "rb");

    for(int z = 0 ; z < a->mRecCount; z++) {
        fseek(f, a->h_columns_int[colname][z] * len, SEEK_SET);
        fread(a->h_columns_char[colname] + z*len, 1, len, f);
    };
    fclose(f);

    if(!host) {
        void *d;
        hipMalloc((void **) &d, a->mRecCount*len);
        a->d_columns_char[colname] = (char*)d;

        hipMemcpy(a->d_columns_char[colname], a->h_columns_char[colname], len*a->mRecCount, hipMemcpyHostToDevice);

	    if (ord.compare("DESC") == 0 )
			str_sort(a->d_columns_char[colname], a->mRecCount, raw_ptr, 1, len);
		else
			str_sort(a->d_columns_char[colname], a->mRecCount, raw_ptr, 0, len);

        hipFree(d);
    }
    else {
	    if (ord.compare("DESC") == 0 )
			str_sort_host(a->h_columns_char[colname], a->mRecCount, raw_ptr, 1, len);
		else
			str_sort_host(a->h_columns_char[colname], a->mRecCount, raw_ptr, 0, len);
    };
}



size_t getTotalSystemMemory()
{
    long pages = sysconf(_SC_PHYS_PAGES);
    long page_size = sysconf(_SC_PAGE_SIZE);
    return pages * page_size;
}
*/

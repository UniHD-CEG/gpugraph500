#include "hip/hip_runtime.h"
#include "../comp_opt.h"
#include "cuda_bfs.h"
#include "b40c/util/error_utils.cuh"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <cstdlib>
#include <algorithm>
#include <functional>

#ifdef _COMPRESSION
#include "../types_bfs.h"
#endif

#if defined( __PMODE__)
#include <parallel/algorithm>
#endif

CUDA_BFS::CUDA_BFS(MatrixT &_store, int &num_gpus, double _queue_sizing,
                   int64_t _verbosity
                  ) :
    GlobalBFS
    <CUDA_BFS, vertexType, unsigned char, MatrixT>(_store),
    verbosity(_verbosity),
    queue_sizing(_queue_sizing),
    vmask(0)
#ifdef _DEBUG
    , checkQueue(0, _store.getLocRowLength(), 0, _store.getLocColLength())
#endif
{
    int cpro_verbosity;
    b40c::util::B40CPerror(hipSetDeviceFlags(hipDeviceMapHost),
                           "Enabling of the allocation of pinned host memory failed", __FILE__, __LINE__);

    if (num_gpus == 0)
    {
        b40c::util::B40CPerror(hipGetDeviceCount(&num_gpus),
                               "Can't get number of devices!", __FILE__, __LINE__);
    }

    //expect symmetries
    if (store.getNumRowSl() != store.getNumColumnSl())
    {
        printf("Partitioning has to be symmetric.\n");
        exit(1);
    }
    predecessor = new vertexType[store.getLocColLength()];

#ifdef _COMPRESSION
    fq_tp_typeC = MPIcompressed;
    fq_tp_type = MPI_INT64_T;
#else
    fq_tp_type = MPI_INT64_T;
#endif

    bm_type = MPI_UNSIGNED_CHAR;
    fq_64_length = static_cast<vertexType>(std::max(store.getLocRowLength(), store.getLocColLength()) * queue_sizing);
    //fq_64 = new vertexType[fq_64_length];
    hipHostAlloc(&fq_64, fq_64_length * sizeof(vertexType), hipHostMallocDefault);
    //multipurpose buffer
    qb_length = 0ULL;
    hipHostAlloc(&queuebuff, fq_64_length * sizeof(vertexType), hipHostMallocDefault);
    rb_length = 0ULL;
    hipHostAlloc(&redbuff, fq_64_length * sizeof(vertexType), hipHostMallocDefault);

    csr_problem = new Csr;

#ifdef INSTRUMENTED
    bfsGPU = new EnactorMultiGpu<Csr, true>;
#else
    bfsGPU = new EnactorMultiGpu<Csr, false>;
#endif

    cpro_verbosity = 0ULL;
    if (verbosity >= 24ULL)
    {
        cpro_verbosity = 2ULL;
    }
    else if (verbosity >= 8ULL)
    {
        cpro_verbosity = 1ULL;
    }
    b40c::util::B40CPerror(csr_problem->FromHostProblem(
                               false,                  //bool          stream_from_host,
                               store.getLocRowLength(),//SizeT         nodes,
                               store.getEdgeCount(),   //SizeT         edges,
                               store.getColumnIndex(), //VertexId      *h_column_indices,
                               store.getRowPointer(),  //SizeT         *h_row_offsets,
                               num_gpus,               //int       num_gpus,
                               cpro_verbosity          //verbosity
                           ), "FromHostProblem failed!", __FILE__, __LINE__);

    // Enable symmetric peer access between gpus
    // from test_bfs.cu
    // if(peerPossible)
    Csr::GraphSlice *gs;
    Csr::GraphSlice *gs_other;
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        gs = csr_problem->graph_slices[gpu];
        for (int other_gpu = (gpu + 1) % num_gpus; other_gpu != gpu; other_gpu = (other_gpu + 1) % num_gpus)
        {
            gs_other = csr_problem->graph_slices[other_gpu];
            // Set device
            if (b40c::util::B40CPerror(hipSetDevice(gs->gpu),
                                       "MultiGpuBfsEnactor hipSetDevice failed", __FILE__, __LINE__))
            {
                exit(1);
            }
            hipError_t error = hipDeviceEnablePeerAccess(gs_other->gpu, 0);
            if ((error != hipSuccess) && (error != hipErrorPeerAccessAlreadyEnabled))
            {
                b40c::util::B40CPerror(error, "MultiGpuBfsEnactor hipDeviceEnablePeerAccess failed", __FILE__,
                                       __LINE__);
                int canAccessPeer;
                b40c::util::B40CPerror(hipDeviceCanAccessPeer(&canAccessPeer, gs->gpu, gs_other->gpu),
                                       "Can not access device!", __FILE__, __LINE__);
                if (canAccessPeer)
                {
                    fprintf(stderr, "Can access peer %d from %d!\n", gs_other->gpu, gs->gpu);
                }
                else
                {
                    fprintf(stderr, "Can't access peer %d from %d!\n", gs_other->gpu, gs->gpu);
                }
            }
        }
    }
}

CUDA_BFS::~CUDA_BFS()
{
    if (vmask != 0)
    {
        hipHostFree(vmask);
    }

    delete bfsGPU;
    delete csr_problem;
    hipHostFree(redbuff);
    hipHostFree(queuebuff);
    hipHostFree(fq_64);
    delete[] predecessor;
}

/*
 * Performs a memcpy to the FQ variable.
 * FQ variables may require specific device calls.
 */
void CUDA_BFS::bfsMemCpy(vertexType *&dst, vertexType *src, size_t size)
{
    hipMemcpy(dst, src, size * sizeof(vertexType), hipMemcpyHostToHost);
}

/*
 * Function for reduction of the current and incoming frontier queue
 * Supports now only one gpu, because the vertexranges are not continuous
 */
void CUDA_BFS::reduce_fq_out(vertexType globalstart, long size, vertexType *startaddr, int insize)
{

#ifdef _DEBUG
    CheckQueue<vertexType>::ErrorCode errorCode;
    if ((errorCode = checkQueue.checkCol(startaddr, insize)) != CheckQueue<vertexType>::ErrorCode::Valid)
    {
        std::cerr << "(" << store.getLocalRowID() << ":" << store.getLocalColumnID() << ") ";
        switch (errorCode)
        {
        case CheckQueue<vertexType>::ErrorCode::InvalidLength: std::cerr << "Recieved queue with invalid length";
            break;
        case CheckQueue<vertexType>::ErrorCode::IdsOutOfRange: std::cerr << "Recieved queue with ids out of range";
            break;
        case CheckQueue<vertexType>::ErrorCode::NotSorted: std::cerr << "Recieved not sorted queue";
            break;
        case CheckQueue<vertexType>::ErrorCode::DuplicteIds: std::cerr << "Recieved queue with duplicate ids";
            break;
        default: std::cerr << "Recieved invalid queue";
        }
        std::cerr << " from source node." << std::endl;
    }
#endif

    typename MatrixT::vertexType *start_local;
    typename MatrixT::vertexType *end_local;
    typename MatrixT::vertexType *endofresult;

    // determine the local range for the reduction
    start_local = std::lower_bound(queuebuff, queuebuff + qb_length, globalstart,
    [](vertexType a, vertexType b) { return a < (b & Csr::ProblemType::VERTEX_ID_MASK); });
    end_local = std::upper_bound(start_local, queuebuff + qb_length, globalstart + size - 1,
    [](vertexType a, vertexType b) { return b > (a & Csr::ProblemType::VERTEX_ID_MASK); });
    //reduction
    endofresult = std::set_union(start_local, end_local, startaddr, startaddr + insize, redbuff);

#ifdef _DEBUG
    //CheckQueue<vertexType>::ErrorCode errorCode;
    if ((errorCode = checkQueue.checkCol(redbuff, endofresult - redbuff)) != CheckQueue<vertexType>::ErrorCode::Valid)
    {
        std::cerr << "(" << store.getLocalRowID() << ":" << store.getLocalColumnID() << ") ";
        switch (errorCode)
        {
        case CheckQueue<vertexType>::ErrorCode::InvalidLength:
            std::cerr << "Try to send queue with invalid length to the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::IdsOutOfRange:
            std::cerr << "Try to send queue with ids out of range to the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::NotSorted:
            std::cerr << "Try to send not sorted queue to the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::DuplicteIds:
            std::cerr << "Try to send queue with duplicate ids to the device." << std::endl;
            break;
        default:
            std::cerr << "Try to send invalid queue to the device." << std::endl;
        }
    }
#endif

    qb_length = endofresult - redbuff;
    std::swap(queuebuff, redbuff);
}

void CUDA_BFS::getOutgoingFQ(vertexType *&startaddr, int &outsize)
{
    startaddr = queuebuff;
    outsize = qb_length;
}

/*
 * -set the Outgoing queue after the column reduction
 * -recompute the visited mask
 */
void CUDA_BFS::setModOutgoingFQ(vertexType *startaddr, int insize)
{

    const int numGpus = csr_problem->num_gpus;

// #ifdef _CUDA_OPENMP
//     #pragma omp parallel for
// #endif

    for (int i = 0; i < numGpus; ++i)
    {
        Csr::GraphSlice *gs = csr_problem->graph_slices[i];
        b40c::util::B40CPerror(hipStreamSynchronize(gs->stream),
                               "Can't synchronize Stream.", __FILE__, __LINE__);
    }

    if (startaddr != 0)
    {
        std::swap(fq_64, queuebuff);
        qb_length = insize;
    }
    //update visited
    for (uint64_t i = 0; i < qb_length; ++i)
    {
        typename Csr::ProblemType::VertexId vtxID = queuebuff[i] & Csr::ProblemType::VERTEX_ID_MASK;
        vmask[vtxID >> 3] |= 1 << (vtxID & 0x7);
    }

    int visited_mask_bytes;
    for (int i = 0; i < numGpus; ++i)
    {
        typename Csr::GraphSlice *gs = csr_problem->graph_slices[i];
        visited_mask_bytes = ((csr_problem->nodes * sizeof(typename Csr::VisitedMask)) + 8 - 1) / 8;
        b40c::util::B40CPerror(hipMemcpyAsync(gs->d_visited_mask,
                                               vmask,
                                               visited_mask_bytes,
                                               hipMemcpyHostToDevice,
                                               gs->stream
                                              ), "Copy of d_filer_mask to device failed", __FILE__, __LINE__);
    }
}

/*
 *  Expect symmetric partitioning
 */
void CUDA_BFS::getOutgoingFQ(vertexType globalstart, long size, vertexType *&startaddr, int &outsize)
{
    typename MatrixT::vertexType *start_local;
    typename MatrixT::vertexType *end_local;

    // determine the local range for the reduction
    start_local = std::lower_bound(queuebuff, queuebuff + qb_length, globalstart,
    [](vertexType a, vertexType b) { return a < (b & Csr::ProblemType::VERTEX_ID_MASK); });
    end_local = std::upper_bound(start_local, queuebuff + qb_length, globalstart + size - 1,
    [](vertexType a, vertexType b) { return b > (a & Csr::ProblemType::VERTEX_ID_MASK); });

#ifdef _DEBUG
    CheckQueue<vertexType>::ErrorCode errorCode;
    if ((errorCode = checkQueue.checkCol(start_local, end_local - start_local)) != CheckQueue<vertexType>::ErrorCode::Valid)
    {
        std::cerr << "(" << store.getLocalRowID() << ":" << store.getLocalColumnID() << ") ";
        switch (errorCode)
        {
        case CheckQueue<vertexType>::ErrorCode::InvalidLength: std::cerr << "Select queue with invalid length";
            break;
        case CheckQueue<vertexType>::ErrorCode::IdsOutOfRange: std::cerr << "Select queue with ids out of range";
            break;
        case CheckQueue<vertexType>::ErrorCode::NotSorted: std::cerr << "Select not sorted queue";
            break;
        case CheckQueue<vertexType>::ErrorCode::DuplicteIds: std::cerr << "Select queue with duplicate ids";
            break;
        default: std::cerr << "Select invalid queue";
        }
        std::cerr << "." << std::endl;
    }
#endif

    startaddr = start_local;
    outsize = end_local - start_local;
}

/*  Sets the incoming FQ.
 *  Expect symmetric partitioning, so all parameters are ignored.
 */
void CUDA_BFS::setIncommingFQ(vertexType globalstart, long size, vertexType *startaddr, int &insize_max)
{
    if (startaddr == fq_64)
    {
        std::swap(fq_64, queuebuff);
    }
    qb_length = insize_max;
}

bool CUDA_BFS::istheresomethingnew()
{
    return !done;
}

void CUDA_BFS::getBackPredecessor()
{
    //terminate all operations


    bfsGPU->testOverflow(*csr_problem);
    b40c::util::B40CPerror(csr_problem->ExtractResults(predecessor, store.localtoglobalRow(0)),
                           "Extraction of result failed", __FILE__, __LINE__);
    bfsGPU->finalize();
    const int64_t sizeOfMType = 8LL * sizeof(MType);
    const int64_t storeColLength = (int64_t)store.getLocColLength();

#ifdef _CUDA_OPENMP
    #pragma omp parallel
    {
        #pragma omp for schedule (guided, 2)
#endif

        for (int64_t i = 0LL; i < mask_size; ++i)
        {
            MType tmp = 0;
            const int64_t isize = i * sizeOfMType;
            for (int64_t j = 0LL; j < sizeOfMType; ++j)
            {
                const int64_t jsize = isize + j;
                const vertexType pred = predecessor[jsize];
                if ((pred != -1) && ((jsize) < storeColLength))
                {
                    tmp |= 1 << j;
                    if (pred > -2)
                    {
                        predecessor[jsize] = store.localtoglobalRow(
                                                 pred & Csr::ProblemType::VERTEX_ID_MASK);
                    }
                    else
                    {
                        predecessor[jsize] = store.localtoglobalCol(jsize);
                    }
                }
            }
            owenmask[i] = tmp;
        }

#ifdef _CUDA_OPENMP
    }
#endif
}

void CUDA_BFS::getBackOutqueue()
{
    long queue_sizes[csr_problem->num_gpus];
    const int numGpus = csr_problem->num_gpus;

#ifdef _DEBUG
    b40c::util::B40CPerror(bfsGPU->testOverflow(*csr_problem));
#endif

    //get length of next queues
#ifdef _CUDA_OPENMP
    #pragma omp parallel for
#endif

    for (int i = 0; i < numGpus; ++i)
    {
        Csr::GraphSlice *gs = csr_problem->graph_slices[i];
        queue_sizes[i] = bfsGPU->getQueueSize(gs->gpu, gs->stream);
        b40c::util::B40CPerror(hipStreamSynchronize(gs->stream), "Can't synchronize device.", __FILE__, __LINE__);

    }
    //sort values on the gpu
    for (int i = 0; i < numGpus; ++i)
    {
        typename Csr::GraphSlice *gs = csr_problem->graph_slices[i];
        b40c::util::B40CPerror(hipSetDevice(gs->gpu));
        thrust::device_ptr <typename MatrixT::vertexType> multigpu(gs->frontier_queues.d_keys[0]);
        thrust::sort(multigpu, multigpu + queue_sizes[i]);
    }



    qb_length = 0ULL;//csr_problem->num_gpus;
    typename MatrixT::vertexType *qb_nxt = queuebuff;
    // copy next queue to host
    for (int i = 0; i < numGpus; ++i)
    {
        typename Csr::GraphSlice *gs = csr_problem->graph_slices[i];
        b40c::util::B40CPerror(hipStreamSynchronize(gs->stream),
                               "Can't synchronize device.", __FILE__, __LINE__);
        b40c::util::B40CPerror(hipMemcpyAsync(qb_nxt,
                                               gs->frontier_queues.d_keys[0],
                                               queue_sizes[i] * sizeof(typename Csr::VertexId),
                                               hipMemcpyDeviceToHost,
                                               gs->stream
                                              ), "Copy of d_keys[0] failed", __FILE__, __LINE__);
        qb_nxt += queue_sizes[i];
        qb_length += queue_sizes[i];
    }

    //#pragma omp parallel for
    for (int i = 0; i < numGpus; ++i)
    {
        Csr::GraphSlice *gs = csr_problem->graph_slices[i];
        b40c::util::B40CPerror(hipStreamSynchronize(gs->stream),
                               "Can't synchronize Stream.", __FILE__, __LINE__);
    }

    // Queue preprocessing
    //Uniqueness
    typename MatrixT::vertexType *qb_nxt_in = queuebuff;
    typename MatrixT::vertexType *qb_nxt_out = redbuff;
    for (int i = 0; i < numGpus; ++i)
    {
        typename MatrixT::vertexType *start_in = std::upper_bound(qb_nxt_in, qb_nxt_in + queue_sizes[i], -1);
        typename MatrixT::vertexType *end_out = std::unique_copy(start_in, qb_nxt_in + queue_sizes[i], qb_nxt_out);
        qb_nxt_in += queue_sizes[i];
        qb_nxt_out = end_out;
    }
    qb_length = qb_nxt_out - redbuff;
    std::swap(queuebuff, redbuff);
#ifdef _DEBUG
    CheckQueue<vertexType>::ErrorCode errorCode;
    if ((errorCode = checkQueue.checkCol(queuebuff, qb_length)) != CheckQueue<vertexType>::ErrorCode::Valid)
    {
        std::cerr << "(" << store.getLocalRowID() << ":" << store.getLocalColumnID() << ") ";
        switch (errorCode)
        {
        case CheckQueue<vertexType>::ErrorCode::InvalidLength:
            std::cerr << "Got queue with invalid length from the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::IdsOutOfRange:
            std::cerr << "Got queue with ids out of range from the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::NotSorted:
            std::cerr << "Got not sorted queue from the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::DuplicteIds:
            std::cerr << "Got queue with duplicate ids from the device." << std::endl;
            break;
        default:
            std::cerr << "Got invalid queue from the device." << std::endl;
        }
    }
#endif
}

void CUDA_BFS::setBackInqueue()
{
    long queue_sizes[csr_problem->num_gpus];
    typename MatrixT::vertexType *qb_nxt = queuebuff;
    typename MatrixT::vertexType *end_local;
    typename Csr::GraphSlice *gs;
    const int numGpus = csr_problem->num_gpus;

#ifdef _DEBUG
    CheckQueue<vertexType>::ErrorCode errorCode;
    if ((errorCode = checkQueue.checkRow(queuebuff, qb_length)) != CheckQueue<vertexType>::ErrorCode::Valid)
    {
        std::cerr << "(" << store.getLocalRowID() << ":" << store.getLocalColumnID() << ") ";
        switch (errorCode)
        {
        case CheckQueue<vertexType>::ErrorCode::InvalidLength:
            std::cerr << "Try to copy queue with invalid length to the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::IdsOutOfRange:
            std::cerr << "Try to copy queue with ids out of range to the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::NotSorted:
            std::cerr << "Try to copy not sorted queue to the device." << std::endl;
            break;
        case CheckQueue<vertexType>::ErrorCode::DuplicteIds:
            std::cerr << "Try to copy queue with duplicate ids to the device." << std::endl;
            break;
        default:
            std::cerr << "Try to copy invalid queue to the device." << std::endl;
        }
    }
#endif

    // copy next queue to device
    for (int i = 0; i < numGpus; ++i)
    {
        gs = csr_problem->graph_slices[i];

        //determine end of own slice
        end_local = std::upper_bound(qb_nxt, queuebuff + qb_length, gs->gpu,
                                     [](vertexType a, vertexType b)
        {
            return b < ((a & Csr::ProblemType::GPU_MASK) >>
                        Csr::ProblemType::GPU_MASK_SHIFT);
        });
        queue_sizes[i] = end_local - qb_nxt;

        b40c::util::B40CPerror(hipMemcpyAsync(gs->frontier_queues.d_keys[0],
                                               qb_nxt,
                                               queue_sizes[i] * sizeof(typename Csr::VertexId),
                                               hipMemcpyHostToDevice,
                                               gs->stream
                                              ), "Copy of d_keys[0] from device failed", __FILE__, __LINE__);
        qb_nxt = end_local;
    }

    //set length of current queue
#ifdef _CUDA_OPENMP
    #pragma omp parallel for
#endif

    for (int i = 0; i < numGpus; ++i)
    {
        typename Csr::GraphSlice *gs = csr_problem->graph_slices[i];
        bfsGPU->setQueueSize(i, static_cast<typename Csr::SizeT>(queue_sizes[i]), gs->stream);
        b40c::util::B40CPerror(hipStreamSynchronize(gs->stream),
                               "Can't synchronize device.", __FILE__, __LINE__);
    }
}

void CUDA_BFS::setStartVertex(vertexType start)
{
    done = false;
    vertexType src_owner, rstart, lstart = -1;
    typename Csr::GraphSlice *gs;
    int cpro_verbosity = 0, visited_mask_bytes;

    const int numGpus = csr_problem->num_gpus;

#ifdef INSTRUMENTED
    if (verbosity >= 24ULL)
    {
        cpro_verbosity = 2ULL;
    }
    else if (verbosity >= 8ULL)
    {
        cpro_verbosity = 1ULL;
    }
#endif

    if (b40c::util::B40CPerror(csr_problem->Reset(
                                   bfsGPU->GetFrontierType(),
                                   queue_sizing,
                                   cpro_verbosity
                               ), "Reset error.", __FILE__, __LINE__) != hipSuccess)
    {
        MPI_Abort(MPI_COMM_WORLD, 1);
    }
    // Alloc and reset visited mask on host

    gs = csr_problem->graph_slices[0];

    visited_mask_bytes = ((csr_problem->nodes * sizeof(typename Csr::VisitedMask)) + 8 - 1) >> 3;
    if (vmask == 0)
    {
        hipHostAlloc(&vmask, visited_mask_bytes, hipHostMallocDefault);
    }
    std::fill_n(vmask, visited_mask_bytes, 0);

    if (store.isLocalColumn(start))
    {
        lstart = store.globaltolocalCol(start);
        vmask[lstart >> 3] = 1 << (lstart & 0x7);
    }

    //new next queue
    qb_length = 0ULL;

    if (store.isLocalRow(start))
    {
        rstart = store.globaltolocalRow(start);
        src_owner = csr_problem->GpuIndex(rstart);
        rstart |= (src_owner << Csr::ProblemType::GPU_MASK_SHIFT);

        queuebuff[0L] = rstart;
        qb_length = 1ULL;
    }

    if (b40c::util::B40CPerror(bfsGPU->EnactSearch(
                                   *csr_problem,
                                   lstart
                               ), "Start error.", __FILE__, __LINE__) != hipSuccess)
    {
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    for (int i = 0; i < numGpus; ++i)
    {
        //set new visited map
        gs = csr_problem->graph_slices[i];
        b40c::util::B40CPerror(hipMemcpyAsync(gs->d_visited_mask,
                                               vmask,
                                               visited_mask_bytes,
                                               hipMemcpyHostToDevice,
                                               gs->stream
                                              ), "Copy of d_filer_mask from device failed", __FILE__, __LINE__);
        // set new current queue
        if (store.isLocalRow(start) &&
            (((queuebuff[0L] & Csr::ProblemType::GPU_MASK) >> Csr::ProblemType::GPU_MASK_SHIFT) == i))
        {
            b40c::util::B40CPerror(hipMemcpyAsync(gs->frontier_queues.d_keys[0],
                                                   &queuebuff[0L],
                                                   sizeof(typename Csr::VertexId),
                                                   hipMemcpyHostToDevice,
                                                   gs->stream
                                                  ), "Copy of d_keys[0] from device failed", __FILE__, __LINE__);
            bfsGPU->setQueueSize(i, static_cast<typename Csr::SizeT>(1), gs->stream);
        }
        else
        {
            bfsGPU->setQueueSize(i, static_cast<typename Csr::SizeT>(0), gs->stream);
        }
    }
}

void CUDA_BFS::runLocalBFS()
{
    const int numGpus = csr_problem->num_gpus;

    //finish outstanding copys
    for (int i = 0; i < numGpus; ++i)
    {
        hipStreamSynchronize(csr_problem->graph_slices[i]->stream);
    }
    //enact expansion kernel
    if (b40c::util::B40CPerror(bfsGPU->EnactIteration(
                                   *csr_problem,
                                   done
                               ), "Iteration error.", __FILE__, __LINE__) != hipSuccess)
    {
        MPI_Abort(MPI_COMM_WORLD, 1);
    }
}
